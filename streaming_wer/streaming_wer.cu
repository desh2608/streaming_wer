#include "hip/hip_runtime.h"
#include "streaming_wer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <c10/cuda/CUDAStream.h>



template <typename scalar_t>
__global__ void streaming_levenshtein_distance_kernel(
        const int* __restrict__ source,
        const int* __restrict__ target,
        const int* __restrict__ source_length,
        const int* __restrict__ target_length,
        const float* __restrict__ source_delays,
        const float* __restrict__ target_delays,
        const size_t source_size,
        const size_t target_size,
        const float threshold,
        const int ins_cost, const int del_cost, const int sub_cost, const int str_cost,
        int* __restrict__ operations) {

    extern __shared__ short err[];

    const int i = blockIdx.x;

    auto err_prev = err;
    auto err_curr = err + (target_size + 1);

    const int* hyp_begin = source + i * source_size;
    const int* ref_begin = target + i * target_size;
    const float* hyp_delays_begin = source_delays + i * source_size;
    const float* ref_delays_begin = target_delays + i * target_size;

    int hyp_size = source_length[i];
    int ref_size = target_length[i];

    for (int r = 0; r <= ref_size; ++r) {
        err_prev[r] = r * del_cost; // total_cost = del_cost
    }

    for (int h = 1; h <= hyp_size; ++h) {

        err_curr[0] = err_prev[0] + ins_cost;    // total_cost = ins_cost

        auto hyp = hyp_begin + h - 1;
        auto hyp_delay = hyp_delays_begin + h - 1;

        for (int r = 1; r <= ref_size; ++r) {

            int ins_err = err_prev[r] + ins_cost;
            int del_err = err_curr[r-1] + del_cost;
            int sub_err;

            auto ref = ref_begin + r - 1;
            auto ref_delay = ref_delays_begin + r - 1;

            if (*hyp == *ref) {
                if (*hyp_delay - *ref_delay <= threshold) {
                    sub_err = err_prev[r-1];
                } else {
                    sub_err = err_prev[r-1] + str_cost;
                }
            } else {
                sub_err = err_prev[r-1] + sub_cost;
            }

            if (sub_err < ins_err && sub_err < del_err) {

                err_curr[r] = sub_err;                  // total_cost

            } else if (del_err < ins_err) {

                err_curr[r] = del_err;                  // total_cost

            } else {

                err_curr[r] = ins_err;                  // total_cost
            }

        }

        // alternate for the next recursion
        short* temp = err_prev;
        err_prev = err_curr;
        err_curr = temp;
    }

    operations[i] = err_prev[ref_size];
}



torch::Tensor StreamingLevenshteinDistanceCuda(
        torch::Tensor source,
        torch::Tensor target,
        torch::Tensor source_length,
        torch::Tensor target_length,
        torch::Tensor source_delays,
        torch::Tensor target_delays,
        float threshold,
        int ins_cost, int del_cost, int sub_cost, int str_cost) {

    const auto batch_size = source.size(0);
    const auto shared_size = (target.size(1) + 1) * 2 * sizeof(short);

    at::TensorOptions options(source.device());

    options = options.dtype(at::ScalarType::Int);

    auto operations = torch::empty(batch_size, options);

    auto stream = c10::cuda::getCurrentCUDAStream(source.device().index());

    AT_DISPATCH_ALL_TYPES(source.scalar_type(), "streaming_levenshtein_distance", ([&] {
        streaming_levenshtein_distance_kernel<int><<<batch_size, 1, shared_size, stream>>>(
            source.data<int>(),
            target.data<int>(),
            source_length.data<int>(),
            target_length.data<int>(),
            source_delays.data<float>(),
            target_delays.data<float>(),
            source.size(1),
            target.size(1),
            threshold,
            ins_cost, del_cost, sub_cost, str_cost,
            operations.data<int>());
    }));

    return operations;
}
